#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <type_traits>
#include "tensor_permute_kernel.cuh"
#include "kernel_tools.h"
#include "cudaBasics.h"
#include "cuda_settings.h"
#include "auto_graph.h"

__device__ void __syncthreads();

namespace dnnbasic
{
	using gpuArray = smallGPUArray<uint32_t, tensor<uint32_t>::MAX_DIMENSION_COUNT>;

	static const uint32_t THREADS_PER_BLOCK = 1024;
	static const uint32_t THREADS_PER_WARP = 32;
	static const uint32_t WARPS_PER_BLOCK = THREADS_PER_BLOCK / THREADS_PER_WARP;

	template<typename T>
	__device__ T getWarpSum(const T threadValue)
	{
		T warpSum = threadValue;
		for (uint32_t i = THREADS_PER_WARP / 2; i > 0; i /= 2)
		{
			warpSum += __shfl_down_sync(0xffffffff, warpSum, i);
		}

		return warpSum;
	}

	template<typename T>
	__global__ void sumKernel(
		const cudabasic::span<T> input, 
		cudabasic::span<T> output,
		const uint32_t sumElementStride,
		const uint32_t sumDimSize,
		const uint32_t sumsToMake,
		const uint32_t blocksMade)
	{
		extern __shared__ __align__(sizeof(T)) int8_t sharedArray[];
		T* sharedMemT = reinterpret_cast<T*>(sharedArray);

		const uint32_t sumElemIdx = blockIdx.x * blockDim.x + threadIdx.x;

		for (uint32_t i = blockIdx.y; i < sumsToMake; i += blocksMade)
		{
			//if index is out of bounds then load zero instead
			//as all threads in a warp are needed to sum
			//and keeping all threads to begin with it the
			//easiest way to do that
			const T value = sumElemIdx >= sumDimSize ? 0 : input[sumElemIdx * sumElementStride + (i / sumElementStride) * sumElementStride * sumDimSize + (i % sumElementStride)];

			//Make warp sum
			const T warpSum = getWarpSum(value);

			//First thread in each warp will store their sum
			//in shared memory so the first warp can sum it up
			if (threadIdx.x % THREADS_PER_WARP == 0)
			{
				sharedMemT[threadIdx.x / WARPS_PER_BLOCK] = warpSum;
			}
			__syncthreads();

			//First warp in each block will now
			//make a block sum
			T blockSum = 0;
			if (threadIdx.x < WARPS_PER_BLOCK)
			{
				blockSum = getWarpSum(sharedMemT[threadIdx.x]);
			}
			__syncthreads();

			//First thread in block will now atomic add the result
			if (threadIdx.x == 0)
			{
				if constexpr (std::is_integral<T>::value && std::is_signed<T>::value)
				{
					using unsigned_T = typename std::make_unsigned<T>::type;
					atomicAdd(reinterpret_cast<unsigned_T*>(&output[blockIdx.y]), (unsigned_T)blockSum);
				}
				else
				{
					atomicAdd(&output[blockIdx.y], blockSum);
				}
			}
		}
	}

	template<typename T>
	void tensorSum(const tensor<T>& input, tensor<T>& output, const uint32_t sumDimIdx)
	{
		if constexpr (sizeof(T) < 4)
		{
			throw std::runtime_error("Sum is currently not supported for that tensor type.");
		}
		else
		{
			uint32_t sumElementStride = 1;
			for (size_t i = sumDimIdx + 1; i < input.getDimensions().size(); i++)
			{
				sumElementStride *= input.getDimensions()[i].dim;
			}

			const uint32_t sumDim = input.getDimensions()[sumDimIdx].dim;
			const uint32_t dimsToSum = output.elementCount();
			const uint32_t blocksMade = std::min(dimsToSum, 40u);

			const dim3 blockDim(THREADS_PER_BLOCK);
			const dim3 gridDim(integerCeilDivision(sumDim, blockDim.x), blocksMade);
			if (autoGraph::isRecordingGraph())
			{
				const std::vector<void*> inputPtrs = { reinterpret_cast<void*>(input.getGPUArray().begin()), reinterpret_cast<void*>(output.getGPUArray().begin()) };
				const void* outputPtr = reinterpret_cast<void*>(output.getGPUArray().begin());
				autoGraph::addMemsetNode(outputPtr, outputPtr, output.getGPUArray(), 0);
				autoGraph::addKernelNode(inputPtrs, outputPtr, sumKernel<T>, blockDim, gridDim, (uint32_t)sizeof(T) * WARPS_PER_BLOCK, input.getGPUArrayConst(), output.getGPUArray(), sumElementStride, sumDim, dimsToSum, blocksMade);
			}
			else
			{
				hipMemset(output.getGPUArray().begin(), 0, output.elementCount() * sizeof(T));
				cudabasic::executeKernel(sumKernel<T>, blockDim, gridDim, sizeof(T) * WARPS_PER_BLOCK, cuda::getDefaultStream(), input.getGPUArrayConst(), output.getGPUArray(), sumElementStride, sumDim, dimsToSum, blocksMade);
			}
		}
	}

	template void tensorSum(const tensor<bool>& input, tensor<bool>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<uint8_t>& input, tensor<uint8_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<uint16_t>& input, tensor<uint16_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<uint32_t>& input, tensor<uint32_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<uint64_t>& input, tensor<uint64_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<int8_t>& input, tensor<int8_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<int16_t>& input, tensor<int16_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<int32_t>& input, tensor<int32_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<int64_t>& input, tensor<int64_t>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<float>& input, tensor<float>& output, const uint32_t sumDimIdx);
	template void tensorSum(const tensor<double>& input, tensor<double>& output, const uint32_t sumDimIdx);
}