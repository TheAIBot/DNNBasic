#include "hip/hip_runtime.h"
#include "tensor_kernels.cuh"
#include "cudaBasics.h"

namespace dnnbasic
{
	template<typename T>
	__global__ void multiplyGPU(const cudabasic::span<T> left, const cudabasic::span<T> right, cudabasic::span<T> output)
	{
		const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= left.size())
		{
			return;
		}
		output[index] = left[index] * right[index];
	}

	void tensorMultiply(const tensor<float>& left, const tensor<float>& right, const tensor<float>& result)
	{
		const dim3 blockDim(256);
		const dim3 gridDim((left.elementCount() + (blockDim.x - 1)) / blockDim.x);

		cudabasic::executeKernel(multiplyGPU, blockDim, gridDim, left.getGPUArray(), right.getGPUArray(), result.getGPUArray());
	}
}