#include "hip/hip_runtime.h"
#include "tensor_matrix_kernels.cuh"
#include "cudaBasics.h"
#include "matrix.h"

namespace dnnbasic
{
	template <typename T>
	__global__ void matrixMultiplication(const matrix<T> a, const matrix<T> b, matrix<T> c, const uint32_t num_sub_blocks, const uint32_t blockSize)
	{
		//Define some shared memory for a sub block of matrices A an B
		extern __shared__ __align__(sizeof(T)) int8_t sharedArray[];
		T* sharedMemT = reinterpret_cast<T*>(sharedArray);

		matrix<T> As(sharedMemT, blockSize, blockSize);
		matrix<T> Bs(sharedMemT + blockSize * blockSize, blockSize, blockSize);

		// Block index
		const uint32_t bx = blockIdx.x;
		const uint32_t by = blockIdx.y;
		const uint32_t tx = threadIdx.x;
		const uint32_t ty = threadIdx.y;
		//Running sum of product of A and B matrices
		T Csub = 0;

		//iterate through the number of sub matrices of A and B
		for (uint32_t i = 0; i < num_sub_blocks; i++) {
			const uint32_t a_x = tx + i * blockSize;
			const uint32_t a_y = ty + by * blockSize;
			const uint32_t b_x = tx + bx * blockSize;
			const uint32_t b_y = ty + i * blockSize;

			//a submatrix can lie both inside and outside the bounds of the matrix.
			//We can't load any part that lies outside the bounds so instead 0 is
			//loaded into the submatrix because it doesn't change the result of
			//the sub matrix multiplication.
			As[ty][tx] = a.withinBounds(a_x, a_y) ? a[a_y][a_x] : (T)0;
			Bs[ty][tx] = b.withinBounds(b_x, b_y) ? b[b_y][b_x] : (T)0;

			// Wait untill all threads have loaded their values into shared memory.
			__syncthreads();
			for (uint32_t k = 0; k < blockSize; ++k)
			{
				Csub += As[ty][k] * Bs[k][tx];
			}
			__syncthreads();

		}

		const uint32_t c_x = tx + bx * blockSize;
		const uint32_t c_y = ty + by * blockSize;

		// Write the resulting matrix multiplication into the result matrix if 
		// within bounds.
		if (!c.withinBounds(c_x, c_y))
		{
			return;
		}

		c[c_y][c_x] = Csub;
	}

	/// <summary>
	/// If there is a remainder to the division then it adds 1 to the division result
	/// </summary>
	/// <param name="a">numerator</param>
	/// <param name="b">denominator</param>
	/// <returns></returns>
	int integerCeilDivision(int a, int b)	
	{
		//return (int) math.ceil((float)a / b);
		return (a + (b - 1)) / b;
	}

	template <typename T>
	void tensorMatrixMulInternal(const matrix<T>& left, const matrix<T>& right, matrix<T>& result)
	{
		const int matrixWidth = result.getColumns();
		const int matrixHeight = result.getRows();
		
		const uint32_t blockSize = 32; 
		const dim3 blockDim(blockSize, blockSize);
		const uint32_t sharedMemory = sizeof(T) * blockSize * blockSize * 2;
		const dim3 gridDim(integerCeilDivision(matrixWidth, blockDim.x), integerCeilDivision(matrixHeight, blockDim.y));
		const uint32_t num_sub_blocks = integerCeilDivision(left.getColumns(), blockSize);
		
		cudabasic::executeKernel(matrixMultiplication<T>, blockDim, gridDim, sharedMemory, left, right, result, num_sub_blocks, blockSize);
	}
	void tensorMatrixMul(const matrix<bool>& left, const matrix<bool>& right, matrix<bool>& result){tensorMatrixMulInternal(left, right, result);}
	void tensorMatrixMul(const matrix<uint8_t>& left, const matrix<uint8_t>& right, matrix<uint8_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<uint16_t>& left, const matrix<uint16_t>& right, matrix<uint16_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<uint32_t>& left, const matrix<uint32_t>& right, matrix<uint32_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<uint64_t>& left, const matrix<uint64_t>& right, matrix<uint64_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int8_t>& left, const matrix<int8_t>& right, matrix<int8_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int16_t>& left, const matrix<int16_t>& right, matrix<int16_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int32_t>& left, const matrix<int32_t>& right, matrix<int32_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int64_t>& left, const matrix<int64_t>& right, matrix<int64_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<float>& left, const matrix<float>& right, matrix<float>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<double>& left, const matrix<double>& right, matrix<double>& result) { tensorMatrixMulInternal(left, right, result); }
}