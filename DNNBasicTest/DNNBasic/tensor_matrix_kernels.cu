#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tensor_matrix_kernels.cuh"
#include "cudaBasics.h"
#include "matrix.h"
#include "kernel_tools.h"

namespace dnnbasic
{
	template <typename T>
	__device__ void matMul(const matrix<T> a, const matrix<T> b, matrix<T> c, const uint32_t num_sub_blocks, const uint32_t blockSize, dim3 blockOffset, dim3 threadOffset)
	{
		// Block index
		const uint32_t bx = blockOffset.x;
		const uint32_t by = blockOffset.y;
		const uint32_t tx = threadOffset.x;
		const uint32_t ty = threadOffset.y;
		//Running sum of product of A and B matrices
		T Csub = 0;
		
		// need to fix shared memory offset for multidim matrix multiplication

		//Define some shared memory for a sub block of matrices A an B
		extern __shared__ __align__(sizeof(T)) int8_t sharedArray[];
		T* sharedMemT = reinterpret_cast<T*>(sharedArray);

		matrix<T> As(sharedMemT, blockSize, blockSize);
		matrix<T> Bs(sharedMemT + blockSize * blockSize, blockSize, blockSize);

		//iterate through the number of sub matrices of A and B
		for (uint32_t i = 0; i < num_sub_blocks; i++) {
			const uint32_t a_x = tx + i * blockSize;
			const uint32_t a_y = ty + by * blockSize;
			const uint32_t b_x = tx + bx * blockSize;
			const uint32_t b_y = ty + i * blockSize;

			//a submatrix can lie both inside and outside the bounds of the matrix.
			//We can't load any part that lies outside the bounds so instead 0 is
			//loaded into the submatrix because it doesn't change the result of
			//the sub matrix multiplication.
			As[ty][tx] = a.withinBounds(a_x, a_y) ? a[a_y][a_x] : (T)0;
			Bs[ty][tx] = b.withinBounds(b_x, b_y) ? b[b_y][b_x] : (T)0;

			// change this so that we have min(a height, blocksize) <- is this valid?
			// Wait untill all threads have loaded their values into shared memory.
			__syncthreads();
			for (uint32_t k = 0; k < blockSize; ++k)
			{
				Csub += As[ty][k] * Bs[k][tx];
			}
			__syncthreads();

		}

		const uint32_t c_x = tx + bx * blockSize;
		const uint32_t c_y = ty + by * blockSize;

		// Write the resulting matrix multiplication into the result matrix if 
		// within bounds.
		if (!c.withinBounds(c_x, c_y))
		{
			return;
		}

		c[c_y][c_x] = Csub;
	}

	template <typename T>
	__global__ void matrixMultiplication(const matrix<T> a, const matrix<T> b, matrix<T> c, const uint32_t num_sub_blocks, const uint32_t blockSize)
	{
		matMul(a, b, c, num_sub_blocks, blockSize, blockIdx, threadIdx);
	}


	template <typename T>
	void tensorMatrixMulInternal(const matrix<T>& left, const matrix<T>& right, matrix<T>& result)
	{
		const int matrixWidth = result.getColumns();
		const int matrixHeight = result.getRows();
		
		const uint32_t blockSize = 32; 
		const dim3 blockDim(blockSize, blockSize);
		const uint32_t sharedMemory = sizeof(T) * blockSize * blockSize * 2;
		const dim3 gridDim(integerCeilDivision(matrixWidth, blockDim.x), integerCeilDivision(matrixHeight, blockDim.y));
		const uint32_t num_sub_blocks = integerCeilDivision(left.getColumns(), blockSize);
		
		cudabasic::executeKernel(matrixMultiplication<T>, blockDim, gridDim, sharedMemory, left, right, result, num_sub_blocks, blockSize);
	}
	void tensorMatrixMul(const matrix<bool>& left, const matrix<bool>& right, matrix<bool>& result){tensorMatrixMulInternal(left, right, result);}
	void tensorMatrixMul(const matrix<uint8_t>& left, const matrix<uint8_t>& right, matrix<uint8_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<uint16_t>& left, const matrix<uint16_t>& right, matrix<uint16_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<uint32_t>& left, const matrix<uint32_t>& right, matrix<uint32_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<uint64_t>& left, const matrix<uint64_t>& right, matrix<uint64_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int8_t>& left, const matrix<int8_t>& right, matrix<int8_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int16_t>& left, const matrix<int16_t>& right, matrix<int16_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int32_t>& left, const matrix<int32_t>& right, matrix<int32_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<int64_t>& left, const matrix<int64_t>& right, matrix<int64_t>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<float>& left, const matrix<float>& right, matrix<float>& result) { tensorMatrixMulInternal(left, right, result); }
	void tensorMatrixMul(const matrix<double>& left, const matrix<double>& right, matrix<double>& result) { tensorMatrixMulInternal(left, right, result); }
}