#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include <vector>
#include "tensor_multi_dim_matrix_mul.cuh"
#include "cudaBasics.h"
#include "matrix.h"
#include "tensor.h"
#include "kernel_tools.h"
#include "tensor_matrix_kernels.cuh"

namespace dnnbasic
{
	using gpuArray = smallGPUArray<uint32_t, tensor<uint32_t>::MAX_DIMENSION_COUNT>;

	template <typename T>
	__device__ T max(const T a, const T b) 
	{
		return a > b ? a : b;
	}

	template <typename T>
	__device__ T min(const T a, const T b)
	{
		return a < b ? a : b;
	}

	//template <typename T>
	//__device__ void matMulInternal(const matrix<T> a, const matrix<T> b, matrix<T> c, const uint32_t num_sub_blocks, const uint32_t blockSizeX, const uint32_t blockSizeY, const dim3 blockOffset, const dim3 threadOffset, const uint32_t sharedOffset)
	//{
	//	// Block index
	//	const uint32_t bx = blockOffset.x;
	//	const uint32_t by = blockOffset.y;
	//	const uint32_t tx = threadOffset.x;
	//	const uint32_t ty = threadOffset.y;
	//	//Running sum of product of A and B matrices
	//	T Csub = 0;

	//	// need to fix shared memory offset for multidim matrix multiplication

	//	//Define some shared memory for a sub block of matrices A an B
	//	extern __shared__ __align__(sizeof(T)) int8_t sharedArray[];
	//	T* sharedMemT = reinterpret_cast<T*>(sharedArray);

	//	matrix<T> As(sharedMemT + sharedOffset, blockSizeX, blockSizeY);
	//	matrix<T> Bs(sharedMemT + sharedOffset + blockSizeX * blockSizeY, blockSizeX, blockSizeY);

	//	//iterate through the number of sub matrices of A and B
	//	for (uint32_t i = 0; i < num_sub_blocks; i++) {
	//		const uint32_t a_x = tx + i * blockSizeX;
	//		const uint32_t a_y = ty + by * blockSizeY;
	//		const uint32_t b_x = tx + bx * blockSizeX;
	//		const uint32_t b_y = ty + i * blockSizeY;

	//		//a submatrix can lie both inside and outside the bounds of the matrix.
	//		//We can't load any part that lies outside the bounds so instead 0 is
	//		//loaded into the submatrix because it doesn't change the result of
	//		//the sub matrix multiplication.
	//		As[ty][tx] = a.withinBounds(a_x, a_y) ? a[a_y][a_x] : (T)0;
	//		Bs[ty][tx] = b.withinBounds(b_x, b_y) ? b[b_y][b_x] : (T)0;

	//		// change this so that we have min(a height, blocksize) <- is this valid?
	//		// Wait untill all threads have loaded their values into shared memory.
	//		__syncthreads();
	//		for (uint32_t k = 0; k < blockSizeX; ++k)
	//		{
	//			Csub += As[ty][k] * Bs[k][tx];
	//		}
	//		__syncthreads();

	//	}

	//	const uint32_t c_x = tx + bx * blockSizeX;
	//	const uint32_t c_y = ty + by * blockSizeY;

	//	// Write the resulting matrix multiplication into the result matrix if 
	//	// within bounds.
	//	if (!c.withinBounds(c_x, c_y))
	//	{
	//		return;
	//	}

	//	c[c_y][c_x] = Csub;
	//}

	//template <typename T>
	//__global__ void matrixMultiplication(const matrix<T> a, const matrix<T> b, matrix<T> c, const uint32_t num_sub_blocks, const uint32_t blockSize)
	//{
	//	matMulInternal(a, b, c, num_sub_blocks, blockSize, blockSize, blockIdx, threadIdx, 0);
	//}

	__device__ int icd(int a, int b)
	{
		//return (int) math.ceil((float)a / b);
		return (a + (b - 1)) / b;
	}

	template <typename T>
	__global__ void multiDimMatrixMultiplication(
		const cudabasic::span<T> a, 
		const cudabasic::span<T> b, 
		cudabasic::span<T> c,
		const gpuArray aDimStrides,
		const gpuArray bDimStrides,
		const gpuArray cDimStrides,
		const uint32_t aWidth,
		const uint32_t aHeight,
		const uint32_t bWidth,
		const uint32_t bHeight,
		const uint32_t num_sub_blocks)
	{
		const uint32_t cMatrixWidth = bWidth;
		const uint32_t cMatrixHeight = aHeight;

		hipStream_t stream;
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

		const uint32_t blockSize = min(32u, max(cMatrixWidth, cMatrixHeight));
		const dim3 blockDimq(blockSize, blockSize);
		const dim3 gridDimq(icd(cMatrixWidth, blockDimq.x), icd(cMatrixHeight, blockDimq.y));
		const uint32_t sharedMemory = sizeof(T) * blockDimq.x * blockDimq.y * 2;

		for (size_t qweqwe = 0; ; qweqwe++)
		{
			const uint32_t idx = (threadIdx.x + qweqwe * blockDim.x) * (cMatrixWidth * cMatrixHeight);

			if (idx >= c.size())
			{
				return;
			}


			const uint32_t matrixDimsCount = 2;
			uint32_t index[gpuArray::MAX_LENGTH];
			uint32_t x = idx;
			// make x, y, z, .. indecies
			for (uint32_t i = 0; i < cDimStrides.size(); i++)
			{
				index[i] = x / cDimStrides[i];
				x = x % cDimStrides[i];
			}

			// Convert to matricies
			uint32_t aMatrixIndex = 0;
			uint32_t bMatrixIndex = 0;
			const uint32_t cMatrixIndex = idx;
			for (size_t i = 0; i < cDimStrides.size() - matrixDimsCount; i++)
			{
				aMatrixIndex += index[i] * aDimStrides[i];
				bMatrixIndex += index[i] * bDimStrides[i];
			}

			const matrix<T> aMatrix(&a[aMatrixIndex], aWidth, aHeight);
			const matrix<T> bMatrix(&b[bMatrixIndex], bWidth, bHeight);
			matrix<T> cMatrix(&c[cMatrixIndex], cMatrixWidth, cMatrixHeight);

			matrixMultiplication << <gridDimq, blockDimq, sharedMemory, stream >> > (aMatrix, bMatrix, cMatrix, num_sub_blocks, blockDimq.x);
		}

		hipStreamDestroy(stream);
	}

	template <typename T>
	void tensorMultiDimMatMul(const tensor<T>& a, const tensor<T>& b, const tensor<T>& c)
	{

		std::vector<uint32_t> aDims(c.getDimensions().size());
		std::vector<uint32_t> bDims(c.getDimensions().size());
		std::vector<uint32_t> cDims(c.getDimensions().size());

		int32_t aDimsIdx = (int32_t)a.getDimensions().size() - 1;
		int32_t bDimsIdx = (int32_t)b.getDimensions().size() - 1;
		int32_t cDimsIdx = (int32_t)c.getDimensions().size() - 1;

		// Convert aDims and bDims into a shape tensor in which length of the tensor is
		// the same size as the output c. The ideas is to perform an internal broadcasting of a and b
		// such that these can be multiplied.
		for (int32_t i = (int32_t)c.getDimensions().size() - 1; i >= 0; i--)
		{
			if (aDimsIdx < 0)
			{
				aDims[i] = 1;
			}
			else
			{
				aDims[i] = a.getDimensions()[aDimsIdx].dim;
				aDimsIdx--;
			}
			if (bDimsIdx < 0)
			{
				bDims[i] = 1;
			}
			else
			{
				bDims[i] = b.getDimensions()[bDimsIdx].dim;
				bDimsIdx--;
			}
			cDims[i] = c.getDimensions()[i].dim;
		}

		gpuArray aStrides(aDims.size());
		gpuArray bStrides(bDims.size());
		gpuArray cStrides(cDims.size());

		for (uint32_t i = 0; i < cDims.size(); i++)
		{
			uint32_t aStride = 1;
			uint32_t bStride = 1;
			uint32_t cStride = 1;

			// To get the correct stride when using an array we multiply the following dimensions
			// together such that they correspond to accessing index i of the corresponding matrix 
			// with similar dimensions
			for (uint32_t g = i + 1; g < cDims.size(); g++)
			{
				aStride *= aDims[g];
				bStride *= bDims[g];
				cStride *= cDims[g];
			}
			// if dimension is broadcasted then the stride should be 0 to reuse the same matrix again
			aStrides[i] = aStride * ((aDims[i] == 1 && bDims[i] != 1) ? 0 : 1);
			bStrides[i] = bStride * ((bDims[i] == 1 && aDims[i] != 1) ? 0 : 1);
			cStrides[i] = cStride;
		}

		// height and width of the matrix
		const uint32_t aWidth = aDims[aDims.size() - 1];
		const uint32_t bWidth = bDims[bDims.size() - 1];
		const uint32_t aHeight = aDims[aDims.size() - 2];
		const uint32_t bHeight = bDims[bDims.size() - 2];

		const uint32_t blockSize = 784;
		const dim3 blockDim(blockSize);
		const dim3 gridDim(1);
		const uint32_t num_sub_blocks = integerCeilDivision(aWidth, blockSize);

		cudabasic::executeKernel(multiDimMatrixMultiplication<T>, blockDim, gridDim, a.getGPUArrayConst(), b.getGPUArrayConst(), c.getGPUArray(),
			aStrides, bStrides, cStrides, aWidth, aHeight, bWidth, bHeight, num_sub_blocks);
	}
	void tensorMultiDimMatrixMul(const tensor<bool>& a, const tensor<bool>& b, const tensor<bool>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<uint8_t>& a, const tensor<uint8_t>& b, tensor<uint8_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<uint16_t>& a, const tensor<uint16_t>& b, tensor<uint16_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<uint32_t>& a, const tensor<uint32_t>& b, tensor<uint32_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<uint64_t>& a, const tensor<uint64_t>& b, tensor<uint64_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<int8_t>& a, const tensor<int8_t>& b, tensor<int8_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<int16_t>& a, const tensor<int16_t>& b, tensor<int16_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<int32_t>& a, const tensor<int32_t>& b, tensor<int32_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<int64_t>& a, const tensor<int64_t>& b, tensor<int64_t>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<float>& a, const tensor<float>& b, tensor<float>& c) { tensorMultiDimMatMul(a, b, c); }
	void tensorMultiDimMatrixMul(const tensor<double>& a, const tensor<double>& b, tensor<double>& c) { tensorMultiDimMatMul(a, b, c); }
}